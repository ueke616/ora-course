// compile with: nvcc -Xcompiler -fopenmp -o t5 t5.cu -O3 -lineinfo

#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <vector>
using namespace std;

#define cudaCheckErrors(msg) \
  do { \
    hipError_t __err = hipGetLastError(); \
    if (__err != hipSuccess) { \
        fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
            msg, hipGetErrorString(__err), \
            __FILE__, __LINE__); \
        fprintf(stderr, "*** FAILED - ABORTING\n"); \
        exit(1); \
    } \
  } while (0)


#include <sys/time.h>
#define USECPSEC 1000000ULL

// 计时函数，用于计算时间差，单位为微秒
unsigned long long dtime_usec(unsigned long long start){

  timeval tv;
  gettimeofday(&tv, 0);
  return ((tv.tv_sec*USECPSEC)+tv.tv_usec)-start;
}

// perform vector averaging over M vectors of length L,  followed by matrix-vector multiply
// repeat the above N times
// input vectors are stored as a set of N column-major matrices
// for each k in N: output[k] = matrix*input[k]
// cpu 版本的向量平均和矩阵-向量乘法
template<typename T>
void cpu_version1(T *input, T *output, T *matrix, int L, int M, int N){
#pragma omp parallel for
  for (int k = 0; k < N; k++){      // repeat the following, N times
    std::vector<T> v1(L);           // vector length of L
    for (int i = 0; i < M; i++)     // compute average vector over M input vectors
      for (int j = 0; j < L; j++)
        v1[j] += input[k*M*L+j*M+i];
    for (int j = 0; j < L; j++)
      v1[j] /= M;
    for (int i = 0; i < L; i++)     // matrix-vector multiply
      for (int j = 0; j < L; j++)
        output[i*N+k] += matrix[i*L+j]*v1[j];
  }
}

const int my_L = 1024;
const int my_M = 1024;
const int my_N = 1024;

// GPU 版本的向量平均和矩阵-向量乘法
template<typename T>
__global__ void gpu_version1(const T * __restrict__ input, T * __restrict__ output, const T * __restrict__ matrix, const int L, const int M, const int N){

  __shared__ T smem[my_L];
  size_t idx = ((size_t)blockIdx.x)*blockDim.x + threadIdx.x;
  for (int k = 0; k < N; k++){  // iterate over N data sets
    T v1 = 0;
    for (int i = 0; i < M; i++) // perform vector averaging
      v1 += input[k*M*L+idx*M+i];
    v1 /= M;
    for (int i = 0; i < L; i++){ // perform matrix-vector multiply
      __syncthreads();
      smem[threadIdx.x] = v1 * matrix[i*L+idx];
      for (int s = blockDim.x>>1; s > 0; s>>=1){
        __syncthreads();
        if (threadIdx.x < s) smem[threadIdx.x] += smem[threadIdx.x+s];}
      if (!threadIdx.x) output[k+i*N] = smem[0];}
  }
}

// gpu_version2 使得每个块专门处理一个数据集，而 gpu_version1 则要求线程跨数据集进行操作，这可能影响性能和效率，特别是在大规模数据处理时
template<typename T>
__global__ void gpu_version2(const T * __restrict__ input, T * __restrict__ output, const T * __restrict__ matrix, const int L, const int M, const int N){
    __shared__ T smem[my_L];
    size_t idx = threadIdx.x;
    size_t k = blockIdx.x;

    T v1 = 0;
    for (int i = 0; i < M; i++) // perform vector averaging
        v1 += input[k*M*L+idx*M+i];
        v1 /= M;
    for (int i = 0; i < L; i++){ // perform matrix-vector multiply
        __syncthreads();
        smem[threadIdx.x] = v1 * matrix[i*L+idx];
        for (int s = blockDim.x>>1; s > 0; s>>=1){
            __syncthreads();
            if (threadIdx.x < s) 
                smem[threadIdx.x] += smem[threadIdx.x+s];
        }
        if (!threadIdx.x) 
            output[k+i*N] = smem[0];
    }
}

typedef float ft;

int main(){
  ft *d_input, *h_input, *d_output, *h_outputc, *h_outputg, *d_matrix, *h_matrix;
  int L = my_L; int M = my_M; int N = my_N;
  // host allocations
  h_input   = new ft[N*L*M];
  h_matrix  = new ft[L*L];
  h_outputg = new ft[N*L];
  h_outputc = new ft[N*L];
  // data initialization
  for (int i = 0; i < N*L*M; i++) h_input[i] = (rand()&1)+1;  // 1 or 2
  for (int i = 0; i < L*L; i++) h_matrix[i]  = (rand()&1)+1;  // 1 or 2
  // create result to test for correctness
  unsigned long long dt = dtime_usec(0);
  cpu_version1(h_input, h_outputc, h_matrix, L, M, N);
  dt = dtime_usec(dt);
  std::cout << "CPU execution time: " << dt/(float)USECPSEC << "s" << std::endl;
  // device allocations
  hipMalloc(&d_input, N*L*M*sizeof(ft));
  hipMalloc(&d_output,  N*L*sizeof(ft));
  hipMalloc(&d_matrix,  L*L*sizeof(ft));
  cudaCheckErrors("hipMalloc failure");
  // copy input data from host to device
  hipMemcpy(d_input,  h_input,  N*L*M*sizeof(ft), hipMemcpyHostToDevice);
  hipMemcpy(d_matrix, h_matrix,   L*L*sizeof(ft), hipMemcpyHostToDevice);
  hipMemset(d_output, 0, N*L*sizeof(ft));
  cudaCheckErrors("cudaMemcpy/Memset failure");
  // run on device and measure execution time
  dt = dtime_usec(0);
  gpu_version2<<<N, L>>>(d_input, d_output, d_matrix, L, M, N);
  cudaCheckErrors("kernel launch failure");
  hipDeviceSynchronize();
  cudaCheckErrors("kernel execution failure");
  dt = dtime_usec(dt);
  hipMemcpy(h_outputg, d_output, N*L*sizeof(ft), hipMemcpyDeviceToHost);
  cudaCheckErrors("hipMemcpy failure");
  for (int i = 0; i < N*L; i++) if (h_outputg[i] != h_outputc[i]) {std::cout << "Mismatch at " << i << " was: " << h_outputg[i] << " should be: " << h_outputc[i] << std::endl; return 0;}
  std::cout << "Kernel execution time: " << dt/(float)USECPSEC << "s" << std::endl;
  return 0;
}